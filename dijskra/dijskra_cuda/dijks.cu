#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <cstdlib>
#include <limits>
#include <algorithm>

using namespace std;

const int BLOCK_SIZE = 512;


#define idx(i,j,lda) ( (j) + ((i)*(lda)) )

class mySet
{
private:
	int size = 4000;
	bool N[4000];
	int cnt = 4000;
	
public:
	__device__ mySet(){}
	
	__device__ void init(int s)
	{
		this->cnt = s;
		for (int i = 0; i < s; i++)
		{
			N[i] = true;
		}
	}

	__device__ bool contains(int x)
	{
		return N[x];
	}

	__device__ void insert(int x)
	{
		if (N[x] == true)
			return;
		N[x] = true;
		cnt++;
	}

	__device__ void erase(int x)
	{
		if (N[x] == true)
		{
			N[x] = false;
			cnt--;
		}
	}

	__device__ bool empty()
	{
		return (cnt == 0);
	}

 	__device__ int getCount()
 	{
  	 	return cnt;
  	}
};



__device__ int getGlobalIdx_1D_1D()
{
	return blockIdx.x *blockDim.x + threadIdx.x;
}

__device__ int getGlobalIdx_1D_2D()
{
	return blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
}

__device__ int getGlobalIdx_2D_2D()
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y)
		+ (threadIdx.y * blockDim.x) + threadIdx.x;
	return threadId;
}

__device__ int getGlobalIdx_2D_1D()
{
	int blockId   = blockIdx.y * gridDim.x + blockIdx.x;			 	
	int threadId = blockId * blockDim.x + threadIdx.x; 
	return threadId;
}
__device__ int getGlobalIdx_3D_3D()
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x
		+ gridDim.x * gridDim.y * blockIdx.z;
	int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
		+ (threadIdx.z * (blockDim.x * blockDim.y))
		+ (threadIdx.y * blockDim.x) + threadIdx.x;
	return threadId;
}
//zdroj: http://cs.calvin.edu/courses/cs/374/CUDA/CUDA-Thread-Indexing-Cheatsheet.pdf


__global__ void prepareArray(int vertexCnt, int* d)
{
	int threads = gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
  	int cycleCnt = (vertexCnt / threads > 0 ? vertexCnt / threads : 1);
	for (int cycle = 0; cycle < cycleCnt; cycle++)
	{
		int s = (blockIdx.x * blockDim.x + threadIdx.x) + threads * cycle; // pozice na radku
		if(s >= vertexCnt) 
			return;

		for (int i = 0; i < vertexCnt; i++)
		{
			d[vertexCnt *i+s] = INT_MAX / 2;
		}
	}
}

__global__ void dijsktra( int* __restrict__ edgeMatrix, int vertexCnt, int* d)
{
	int threads = gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
  	int cycleCnt = (vertexCnt / threads > 0 ? vertexCnt / threads : 1);
	for (int cycle = 0; cycle < cycleCnt; cycle++)
	{
		int s = (blockIdx.x * blockDim.x + threadIdx.x) + threads * cycle; // pozice na radku
		if(s >= vertexCnt) 
			return;
			
		mySet N;
		
		N.init(vertexCnt);

		d[s*vertexCnt + s] = 0;
		while (!N.empty())
		{
			int localMin = INT_MAX;

			int cnt = N.getCount();
			int u = 0;
			int j = 0;
			for (int i = 0; i < vertexCnt && j < cnt; i++)
			{
				if (!N.contains(i)) continue;
				if (localMin > d[vertexCnt *i+s])
				{
					localMin = d[vertexCnt *i+s];
					u = i;
				}
				j++;
			}
			N.erase(u);

			for (int i = 0; i < vertexCnt; i++)
			{
				if (i == u || !N.contains(i)) continue;

				if (edgeMatrix[u + i*vertexCnt] > 0)
				{
					int alt = d[vertexCnt *u+s] + edgeMatrix[u + i*vertexCnt];
					atomicMin((d + vertexCnt * i + s), alt);
				}
			}
		}
	}
}



int *readMatrix(const char *path, int &n)
{
	ifstream iFile;
	iFile.open(path);
	if (!iFile.is_open())
		return NULL;

	string line;
	getline(iFile, line);
	n = atoi(line.c_str());
	int *matrix = new int[n * n];
  
	int i;
	for (i = 0; getline(iFile, line); i++)
	{
		size_t endpos = line.find_last_not_of(" \t\r\n");
		if (string::npos != endpos)
			line = line.substr(0, endpos + 1);

		for (int j = 0; j < n; j++)
		{
			size_t pos = line.find_first_of(" ");
			matrix[idx(i,j,n)] = stoi(line, &pos, 10);
			line = line.substr(pos);
			
		}
	}
	iFile.close();
	return matrix;

}

void writeMatrix(const char *path, int n, int *matrix)
{
	ofstream oFile;
	oFile.open(path, fstream::trunc | fstream::out);
	if (oFile.is_open())
	{
		for (int i = 0; i < n; i++)
		{
			for (int j = 0; j < n; j++)
			{
				oFile << matrix[idx(i, j, n)] << (j + 1 < n ? " " : "\n");
			}
		}
	}
	oFile.close();
}

int main(int argc, const char* argv[])
{
	if (argc < 3 || argc > 4)
	{
		cout << "Program takes 2 or 3 parameters (matrix and thread count and optional output file)!\n";
		return 1;
	}

    int threadCnt = atoi(argv[2]);
	
    int stc = 0;
    int *matrix = readMatrix(argv[1], stc);
    // reading input file
    if (matrix == NULL){
         cout << "File doesn't exists" << endl;
         cout << argv[1] << endl;
         return 1;
    }
	hipEvent_t start, stop;
	float elapsedTime;



	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);


	int *cumatrix; 
	int *d;
	hipMalloc((void **)&cumatrix, stc * stc * sizeof(int));
	hipMemcpy(cumatrix, matrix,  stc * stc * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void **)&d, stc * stc * sizeof(int));

	prepareArray<<<stc / BLOCK_SIZE, BLOCK_SIZE>>>(stc, d);
    hipError_t code = hipDeviceSynchronize();  
	if (code != hipSuccess)
	{
		fprintf(stdout, "GPUassert: %s \n", hipGetErrorString(code));
	}

   	dijsktra<<<stc / BLOCK_SIZE, BLOCK_SIZE>>>(cumatrix, stc, d);
    code = hipDeviceSynchronize();  
	if (code != hipSuccess)
	{
		fprintf(stdout, "GPUassert: %s \n", hipGetErrorString(code));
	}
  
	int *outM = new int[stc*stc];

  	hipMemcpy(outM, d,  stc * stc * sizeof(int), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	cout << "Time: " << elapsedTime << endl;
	if (argc == 4)
	{
		cout << "Writing results...\n";
		writeMatrix(argv[3], stc, outM);
	}
	hipFree(cumatrix);
  	hipFree(d);
  	delete [] matrix;
  	delete [] outM;

	cout << "\nDone\n";
    return 0;
}
